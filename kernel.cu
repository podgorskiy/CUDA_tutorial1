
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(float* A) 
{
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;

	float cr = float(j) / gridDim.x / blockDim.x * 3.0f - 2.0f;
	float ci = float(i) / gridDim.y / blockDim.y * 3.0f - 1.5f;
	
	float zr = 0.0f;
	float zi = 0.0f;

	int it = 0;

	for (; it < 255 && (zr * zr + zi * zi) < 10.0f; ++it)
	{
		float new_zr = zr * zr - zi * zi + cr;
		float new_zi = 2.0 * zi * zr + ci;
		zr = new_zr;
		zi = new_zi;
	}

	A[j + i * gridDim.x * blockDim.x] = float(it) / 255.0f;
}

int main()
{
	hipError_t cudaStatus = hipSetDevice(0);

	int size_x = 80;
	int size_y = 40;

	int threads_in_block_x = 8;
	int threads_in_block_y = 8;

	int blocks_x = size_x / threads_in_block_x;
	int blocks_y = size_y / threads_in_block_y;

	float* device_array = nullptr;
	hipMalloc(&device_array, size_x * size_y * sizeof(float));

	kernel<<<dim3(blocks_x, blocks_y), dim3(threads_in_block_x, threads_in_block_y)>>>(device_array);

	float* host_array = new float[size_x * size_y];

	hipMemcpy(host_array, device_array, size_x * size_y * sizeof(float), hipMemcpyDeviceToHost);

	printf("\n");
	for (int i = 0; i < size_y; ++i)
	{
		for (int j = 0; j < size_x; ++j)
		{
			if (host_array[j + i * size_x] > 0.5)
			{
				printf("#");
			}
			else
			{
				printf("*");
			}
		}
		printf("\n");
	}

 	return 0;
}
